#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"

// Empty constructor
CudaMatrixMemory::CudaMatrixMemory(const int rows, const int cols) {
    memory_size = sizeof(float) * rows * cols;
    hipMalloc((void**)device_ptr, memory_size);
};


