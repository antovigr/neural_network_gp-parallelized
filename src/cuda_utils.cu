#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"

CudaThrowError::CudaThrowError(hipError_t error): error(error) {}
void CudaThrowError::throwError(std::string custom_msg) {
    if (error != hipSuccess) {
        std::cerr << custom_msg << hipGetErrorString(error) << std::endl;
    }
}

CudaMatrixMemory::CudaMatrixMemory(const int rows, const int cols) : rows(rows), cols(cols) {
    memory_size = sizeof(float) * rows * cols;
    hipError_t err = hipMalloc((void**)&device_ptr, memory_size);
    if (err != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err) << std::endl;
    }
};

CudaMatrixMemory::~CudaMatrixMemory() {
    hipFree(device_ptr);
}

void CudaMatrixMemory::sendMatrix2Device(const float *carray) {
    hipMemcpy(device_ptr, carray, memory_size, hipMemcpyHostToDevice);
}

void CudaGrid::setKernelGrid(const int blocksize_x, const int blocksize_y, const int rows, const int cols) {
    threads = dim3(blocksize_x, blocksize_y);
    grid = dim3((cols + blocksize_x - 1) / blocksize_x, (rows + blocksize_y - 1) / blocksize_y);
}