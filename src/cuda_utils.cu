#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"

CudaMatrixMemory::CudaMatrixMemory(const int rows, const int cols) : rows(rows), cols(cols) {
    memory_size = sizeof(float) * rows * cols;
    hipError_t err = hipMalloc((void**)&device_ptr, memory_size);
    if (err != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err) << std::endl;
    }
};

CudaMatrixMemory::~CudaMatrixMemory() {
    hipFree(device_ptr);
}

void CudaMatrixMemory::sendMatrix2Device(const float *carray) {
    hipMemcpy(device_ptr, carray, memory_size, hipMemcpyHostToDevice);
}


