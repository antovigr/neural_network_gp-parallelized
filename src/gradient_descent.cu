#include "hip/hip_runtime.h"
#include <iostream>
#include <xtensor/xarray.hpp>
#include <xtensor-blas/xlinalg.hpp>
#include <vector>
#include <string>
#include "gradient_descent.cuh"
#include "utils.hpp"
#include "cuda_utils.cuh"
#include "debugging_utils.hpp"
#include "cuda_operations.cuh"

using namespace std;
using namespace xt;

// Define constructor
// Just init class members
GradientDescent::GradientDescent(const xarray<float> &x_train, const xarray<float> &y_train, vector<xarray<float>> &weights, vector<xarray<float>> &biases, const int batch_size) : x_train(x_train), y_train(y_train), weights(weights), biases(biases), batch_size(batch_size) {
    num_layers = weights.size(); 
    layer_outputs.resize(num_layers);
    layer_activations.resize(num_layers + 1);

    // Initialize cuda arrays (allocate memory)
    
    // Init first layer input, which is the transpose of x_batch
    // Note that the indexing of LA (layer_activations) is somehow décalé: LA_l is the input of the layer L and output of the layer l-1
    int larows = x_train.shape(1);
    int lacols = batch_size;
    CudaMatrixMemory InitLayerActivation(larows, lacols);
    InitLayerActivation.allocateCudaMemory();
    cuda_layer_activations.push_back(InitLayerActivation);
    
    // Init first delta i.e. the delta tensor of the last layer
    int init_deltarows = x_train.shape(1); // nb features
    int init_deltacols = batch_size;
    CudaMatrixMemory InitLayerDelta(init_deltarows, init_deltacols);
    InitLayerDelta.allocateCudaMemory();
    cuda_deltas.push_back(InitLayerDelta);

    for (size_t l = 0; l < num_layers; l++) {
        // Weights
        int wrows = weights[l].shape(0);
        int wcols = weights[l].shape(1);
        CudaMatrixMemory LayerWeights(wrows, wcols);
        LayerWeights.allocateCudaMemory();
        cuda_weights.push_back(LayerWeights);

        // Biases
        int brows = biases[l].shape(0);
        int bcols = biases[l].shape(1);
        CudaMatrixMemory LayerBiases(brows, bcols);
        LayerBiases.allocateCudaMemory();
        cuda_biases.push_back(LayerBiases);

        // Layer output = W_l * LA_l + B_l
        int lorows = wrows;
        int locols = cuda_layer_activations[l].cols;
        CudaMatrixMemory LayerOutput(lorows, locols);
        LayerOutput.allocateCudaMemory();
        cuda_layer_outputs.push_back(LayerOutput);

        // Layer activation = sigmoid( LO_{l-1} )
        // We are pushing the element l + 1 of the vector now (because of the initialization before the loop)
        int larows = lorows;
        int lacols = locols;
        CudaMatrixMemory LayerActivation(larows, lacols);
        LayerActivation.allocateCudaMemory();
        cuda_layer_activations.push_back(LayerActivation);

        if (l > 0) { // Otherwise do nothing since the first value is initialized already
            int deltarows = weights[num_layers - l].shape(1);
            int deltacols = cuda_deltas[num_layers - l].cols;
            CudaMatrixMemory LayerDelta(deltarows, deltacols);
            LayerDelta.allocateCudaMemory();
            cuda_deltas.push_back(LayerDelta);
        }

        printCudaMatrixShapes(LayerWeights, "LayerWeights");
        printCudaMatrixShapes(LayerBiases, "LayerBiases");
        printCudaMatrixShapes(LayerOutput, "LayerOutput");
        printCudaMatrixShapes(LayerActivation, "LayerActivation");
        hipDeviceSynchronize();
    }  
}


// Write gradient descent methods

void GradientDescent::forward_pass(const xarray<float> &x_batch) {

    layer_activations[0] = xt::transpose(x_batch);
    
    // Transform xtarray into carray
    ArrayHandler XBATCH_T;
    XBATCH_T.cast_xtarray(layer_activations[0]);

    // Copy XBATCH_T into cuda_layer_activations[0] i.e. the network's input
    CudaMatrixMemory& network_input = cuda_layer_activations[0];
    network_input.sendMatrix2Device(XBATCH_T.carray);
    
    // Perform computations with cuda
    for (size_t l = 0; l < num_layers; l++) {
        CudaMatrixMemory& w = cuda_weights[l];
        ArrayHandler get_weights;
        get_weights.cast_xtarray(weights[l]);
        w.sendMatrix2Device(get_weights.carray);
        
        CudaMatrixMemory& b = cuda_biases[l];
        ArrayHandler get_biases;
        get_biases.cast_xtarray(biases[l]);
        b.sendMatrix2Device(get_biases.carray);

        CudaMatrixMemory& lo = cuda_layer_outputs[l];
        ArrayHandler get_lo;
        get_lo.cast_xtarray(layer_outputs[l]);
        lo.sendMatrix2Device(get_lo.carray);

        CudaMatrixMemory& la = cuda_layer_activations[l];
        ArrayHandler get_la;
        get_la.cast_xtarray(layer_activations[l]);
        la.sendMatrix2Device(get_la.carray);

        CudaMatrixMemory& la_next = cuda_layer_activations[l + 1];
        ArrayHandler get_la_next;
        get_la_next.cast_xtarray(layer_activations[l + 1]);
        la_next.sendMatrix2Device(get_la_next.carray);
        
        CudaGrid matMulGrid;
        CudaGrid addGrid;
        CudaGrid sigmoidGrid;
        matMulGrid.setKernelGrid(16, 16, w.rows, la.cols);
        addGrid.setKernelGrid(16, 16, w.rows, la.cols);
        sigmoidGrid.setKernelGrid(16, 16, la_next.rows, la_next.cols);

        matrixMulKernel<<<matMulGrid.grid, matMulGrid.threads>>>(w.device_ptr, la.device_ptr, lo.device_ptr, w.rows, w.cols, la.cols); // w * la, write the result in lo
        addBiasToMatrixKernel<<<addGrid.grid, addGrid.threads>>>(lo.device_ptr, b.device_ptr, lo.device_ptr, lo.rows, lo.cols);
        sigmoidKernel<<<sigmoidGrid.grid, sigmoidGrid.threads>>>(lo.device_ptr, la_next.device_ptr, la_next.rows, la_next.cols);

        // Perform computation on CPU
        xarray<float> CPU_lo = xt::linalg::dot(weights[l], layer_activations[l]) + biases[l];
        xarray<float> CPU_la_next = sigmoid(CPU_lo); 
        // Check computation
        checkCudaComputation(lo, CPU_lo, 0.1, "Check computation of layer OUTPUT of l = " + to_string(l));
        checkCudaComputation(la_next, CPU_la_next, 0.1, "Check computation of layer next ACTIVATION of l = " + to_string(l));

        // Copy back the computations into the base pipeline
        float* w_host = w.allocAndSend2Host();
        float* b_host = b.allocAndSend2Host();
        float* lo_host = lo.allocAndSend2Host();
        float* la_host = la.allocAndSend2Host();
        float* la_next_host = la_next.allocAndSend2Host();

        // Assign to base pipeline
        // ArrayHandler lo_xt;
        // lo_xt.cast_carray(lo_host, lo.rows, lo.cols);
        layer_outputs[l] = CPU_lo;
        
        // ArrayHandler la_next_xt;
        // la_next_xt.cast_carray(la_next_host, la_next.rows, la_next.cols);
        layer_activations[l + 1] = CPU_la_next;  
        
        cout << "CPU_lo MATRIX" << endl;
        cout << CPU_lo << endl; 

        print_carray(lo_host, lo.rows, lo.cols, "CUDA_lo_MATRIX");

        delete[] w_host;
        delete[] b_host;
        delete[] lo_host;
        delete[] la_host;
        delete[] la_next_host;
    }

}

void GradientDescent::backward_pass(const xarray<float> &y_batch, const int &current_batch_size, const float &learning_rate) {
    
    vector<xarray<float>> deltas(num_layers);

    // Init delta vector corresponding to the last layer
    xarray<float> &last_activation = layer_activations[num_layers];
    deltas[num_layers - 1] = (last_activation - xt::transpose(y_batch)) * sigmoid_derivative(layer_outputs[num_layers - 1]);

    for (int l = num_layers - 2; l >= 0; l--) {
        deltas[l] = xt::linalg::dot(xt::transpose(weights[l + 1]), deltas[l + 1]) * sigmoid_derivative(layer_outputs[l]);
    }

    // Update weights and biases
    for (int l = 0; l < num_layers; l++) {
        xarray<float> gradient_w = xt::linalg::dot(deltas[l], xt::transpose(layer_activations[l])) / current_batch_size; // Batch size may vary, at the end of epoch
        xarray<float> gradient_b = xt::mean(deltas[l], {1});
        gradient_b = gradient_b.reshape({gradient_b.size(), 1});

        weights[l] -= learning_rate * gradient_w;
        biases[l] -= learning_rate * gradient_b;
    }
}

void GradientDescent::train(const unsigned int &epochs, const float &learning_rate) {
    int dataset_size = x_train.shape()[0];
    int batch_number = (dataset_size / batch_size);

    for (unsigned int epoch = 0; epoch < epochs; epoch++) {

        cout << "Epoch: " << epoch << endl;
        float epoch_mse = 0;
        int batch_id = 0;

        for (int batch_start = 0; batch_start < dataset_size; batch_start += batch_size) {
            if (batch_start + batch_size > x_train.shape(0)) { // Fixed batch size. If the current batch exceeds the end of the dataset, break.
                break;
            }
            
            // Plot currently processed batch number and increment
            cout << "   Batch: " << batch_id << " / " << batch_number << endl;
            batch_id++;

            // Compute the current batch size, as defined normally but smaller if at the end of epoch
            int current_batch_size = batch_size;
            xarray<float> x_batch = xt::view(x_train, range(batch_start, batch_start + current_batch_size), all());
            xarray<float> y_batch = xt::view(y_train, range(batch_start, batch_start + current_batch_size), all());

            // Perform the forward pass
            forward_pass(x_batch); // Modify the layer_activations and layer_outputs
            xarray<float> &last_activation = layer_activations[num_layers];

            // Perform the backward pass
            backward_pass(y_batch,  current_batch_size, learning_rate); // Modify the weights and biases
 
            // Compute the loss for the current batch
            xarray<float> squared_error = xt::pow(last_activation - xt::transpose(y_batch), 2); // Error for each pixel of each observation
            xarray<float> observation_mse = xt::mean(squared_error, {0}); // Mean over all the pixels in the observations
            epoch_mse += xt::sum(observation_mse)() / dataset_size;
        }
        cout << "   MSE: " << epoch_mse << endl;
        loss_history.push_back(epoch_mse);
    }
}

