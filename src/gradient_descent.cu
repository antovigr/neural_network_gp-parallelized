#include "hip/hip_runtime.h"
#include <iostream>
#include <xtensor/xarray.hpp>
#include <xtensor-blas/xlinalg.hpp>
#include <vector>
#include <string>
#include "gradient_descent.cuh"
#include "utils.hpp"
#include "cuda_utils.cuh"
#include "debugging_utils.hpp"
#include "cuda_operations.cuh"

using namespace std;
using namespace xt;

// Define constructor
// Just init class members
GradientDescent::GradientDescent(const xarray<double> &x_train, const xarray<double> &y_train, vector<xarray<double>> &weights, vector<xarray<double>> &biases, const int batch_size) : x_train(x_train), y_train(y_train), weights(weights), biases(biases), batch_size(batch_size) {
    num_layers = weights.size(); 
    layer_outputs.resize(num_layers);
    layer_activations.resize(num_layers + 1);

    // Initialize cuda arrays (allocate memory)
    
    // Init first layer input, which is the transpose of x_batch
    // Note that the indexing of LA (layer_activations) is somehow décalé: LA_l is the input of the layer L and output of the layer l-1
    int larows = x_train.shape(1);
    int lacols = batch_size;
    CudaMatrixMemory InitLayerActivation(larows, lacols);
    InitLayerActivation.allocateCudaMemory();
    cuda_layer_activations.push_back(InitLayerActivation);
    
    for (size_t l = 0; l < num_layers; l++) {
        // Weights
        int wrows = weights[l].shape(0);
        int wcols = weights[l].shape(1);
        CudaMatrixMemory LayerWeights(wrows, wcols);
        LayerWeights.allocateCudaMemory();
        cuda_weights.push_back(LayerWeights);

        // Biases
        int brows = biases[l].shape(0);
        int bcols = biases[l].shape(1);
        CudaMatrixMemory LayerBiases(brows, bcols);
        LayerBiases.allocateCudaMemory();
        cuda_biases.push_back(LayerBiases);

        // Layer output = W_l * LA_l + B_l
        int lorows = wrows;
        int locols = cuda_layer_activations[l].cols;
        CudaMatrixMemory LayerOutput(lorows, locols);
        LayerOutput.allocateCudaMemory();
        cuda_layer_outputs.push_back(LayerOutput);

        // Layer activation = sigmoid( LO_{l-1} )
        // We are pushing the element l + 1 of the vector now (because of the initialization before the loop)
        int larows = lorows;
        int lacols = locols;
        CudaMatrixMemory LayerActivation(larows, lacols);
        LayerActivation.allocateCudaMemory();
        cuda_layer_activations.push_back(LayerActivation);

        printCudaMatrixShapes(LayerWeights, "LayerWeights");
        printCudaMatrixShapes(LayerBiases, "LayerBiases");
        printCudaMatrixShapes(LayerOutput, "LayerOutput");
        printCudaMatrixShapes(LayerActivation, "LayerActivation");
        hipDeviceSynchronize();
    }  
}


// Write gradient descent methods

void GradientDescent::forward_pass(const xarray<double> &x_batch) {

    layer_activations[0] = xt::transpose(x_batch);
    
    // Transform xtarray into carray
    ArrayHandler XBATCH_T;
    XBATCH_T.cast_xtarray(layer_activations[0]);

    // Copy XBATCH_T into cuda_layer_activations[0] i.e. the network's input
    CudaMatrixMemory& network_input = cuda_layer_activations[0];
    network_input.sendMatrix2Device(XBATCH_T.carray);
    
    // Perform computations with cuda
    for (size_t l = 0; l < num_layers; l++) {
        CudaMatrixMemory& w = cuda_weights[l];
        CudaMatrixMemory& b = cuda_biases[l];
        CudaMatrixMemory& lo = cuda_layer_outputs[l];
        CudaMatrixMemory& la = cuda_layer_activations[l];        
        CudaMatrixMemory& la_next = cuda_layer_activations[l + 1];
        
        CudaGrid matMulGrid;
        CudaGrid addGrid;
        CudaGrid sigmoidGrid;
        matMulGrid.setKernelGrid(16, 16, w.rows, la.cols);
        addGrid.setKernelGrid(16, 16, w.rows, la.cols);
        sigmoidGrid.setKernelGrid(16, 16, la_next.rows, la_next.cols);

        matrixMulKernel<<<matMulGrid.grid, matMulGrid.threads>>>(w.device_ptr, la.device_ptr, lo.device_ptr, w.rows, w.cols, la.cols); // w * la, write the result in lo
        addBiasToMatrixKernel<<<addGrid.grid, addGrid.threads>>>(lo.device_ptr, b.device_ptr, lo.device_ptr, lo.rows, lo.cols);
        sigmoidKernel<<<sigmoidGrid.grid, sigmoidGrid.threads>>>(lo.device_ptr, la_next.device_ptr, la_next.rows, la_next.cols);
 
        // Copy back the computations into the base pipeline
        float* w_host = w.allocAndSend2Host();
        float* b_host = b.allocAndSend2Host();
        float* lo_host = lo.allocAndSend2Host();
        float* la_host = la.allocAndSend2Host();
        float* la_next_host = la_next.allocAndSend2Host();

        // Assign to base pipeline
        ArrayHandler lo_xt;
        lo_xt.cast_carray(lo_host, lo.rows, lo.cols);
        layer_outputs[l] = lo_xt.xtarray;
        
        ArrayHandler la_next_xt;
        la_next_xt.cast_carray(la_next_host, la_next.rows, la_next.cols);
        layer_activations[l + 1] = la_next_xt.xtarray;  
    }


    // for (size_t l = 0; l < num_layers; l++) {
    //     layer_outputs[l] = xt::linalg::dot(weights[l], layer_activations[l]) + biases[l];
    //     layer_activations[l + 1] = sigmoid(layer_outputs[l]); // sigmoid is defined in utils/utils.cpp
    // }
}

void GradientDescent::backward_pass(const xarray<double> &y_batch, const int &current_batch_size, const float &learning_rate) {
    
    vector<xarray<double>> deltas(num_layers);

    // Init delta vector corresponding to the last layer
    xarray<double> &last_activation = layer_activations[num_layers];
    deltas[num_layers - 1] = (last_activation - xt::transpose(y_batch)) * sigmoid_derivative(layer_outputs[num_layers - 1]);

    for (int l = num_layers - 2; l >= 0; l--) {
        deltas[l] = xt::linalg::dot(xt::transpose(weights[l + 1]), deltas[l + 1]) * sigmoid_derivative(layer_outputs[l]);
    }

    // Update weights and biases
    for (int l = 0; l < num_layers; l++) {
        xarray<double> gradient_w = xt::linalg::dot(deltas[l], xt::transpose(layer_activations[l])) / current_batch_size; // Batch size may vary, at the end of epoch
        xarray<double> gradient_b = xt::mean(deltas[l], {1});
        gradient_b = gradient_b.reshape({gradient_b.size(), 1});

        weights[l] -= learning_rate * gradient_w;
        biases[l] -= learning_rate * gradient_b;
    }
}

void GradientDescent::train(const unsigned int &epochs, const float &learning_rate) {
    int dataset_size = x_train.shape()[0];
    int batch_number = (dataset_size / batch_size);

    for (unsigned int epoch = 0; epoch < epochs; epoch++) {

        cout << "Epoch: " << epoch << endl;
        float epoch_mse = 0;
        int batch_id = 0;

        for (int batch_start = 0; batch_start < dataset_size; batch_start += batch_size) {
            if (batch_start + batch_size > x_train.shape(0)) { // Fixed batch size. If the current batch exceeds the end of the dataset, break.
                break;
            }
            
            // Plot currently processed batch number and increment
            cout << "   Batch: " << batch_id << " / " << batch_number << endl;
            batch_id++;

            // Compute the current batch size, as defined normally but smaller if at the end of epoch
            int current_batch_size = batch_size;
            xarray<double> x_batch = xt::view(x_train, range(batch_start, batch_start + current_batch_size), all());
            xarray<double> y_batch = xt::view(y_train, range(batch_start, batch_start + current_batch_size), all());

            // Perform the forward pass
            forward_pass(x_batch); // Modify the layer_activations and layer_outputs
            xarray<double> &last_activation = layer_activations[num_layers];

            // Perform the backward pass
            backward_pass(y_batch,  current_batch_size, learning_rate); // Modify the weights and biases
 
            // Compute the loss for the current batch
            xarray<double> squared_error = xt::pow(last_activation - xt::transpose(y_batch), 2); // Error for each pixel of each observation
            xarray<double> observation_mse = xt::mean(squared_error, {0}); // Mean over all the pixels in the observations
            epoch_mse += xt::sum(observation_mse)() / dataset_size;
        }
        cout << "   MSE: " << epoch_mse << endl;
        loss_history.push_back(epoch_mse);
    }
}

